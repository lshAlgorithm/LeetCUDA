#include "hip/hip_runtime.h"
template <const int BM = 128, const int BN = 128, const int BK = 8, const int TM = 8, const int TN = 8>
__global__ void hgemm_t_8x8_slice_k_fp16x4_kernel(half* a, half* b, half* c, int M, int N , int K) { // fpx4: precision in every thread is fp16, while x4 means every thread works for 4 data
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tid = tx + ty * blockDim.x;
    __shared__ half s_a[BM][BK], s_b[BK][BN];
    
    // load data to smem
    int load_smem_a_m = tid / 2;
    int load_smem_a_k = (tid % 2 == 0) ? 0: 4;

    int load_smem_b_k = tid / 32;
    int load_smem_b_n = (tid % 32 == 0) ? 0: 4;

    int load_gmem_a_m = by * BM + load_smem_a_m;
    int load_gmem_b_n = bx * BN + load_smem_b_n;

    if (load_gmem_a_m >= M || load_gmem_b_n >= N) return;

    // for every thread, add up calculation density
    half r_c[TM][TN] = {__float2half(.0f)};

    for (int bk = 0; bk < (K + BK - 1) / BK; ++bk) {
        int load_gmem_a_k = bk * BK + load_smem_a_k;
        int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k; // fuck!
        //...
    }
}